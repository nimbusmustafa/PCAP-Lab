#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  // Matrix size N x N
#define BLOCK_SIZE 16  // Block size (BLOCK_SIZE x BLOCK_SIZE)

// CUDA Kernel for Matrix Multiplication
__global__ void matrixMulKernel(float* A, float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Row of C to compute
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Column of C to compute

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);
    
    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize matrices A and B
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy matrices A and B to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch the matrix multiplication kernel
    matrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Copy result matrix back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Optionally print part of the result
    printf("C[0][0] = %f\n", h_C[0]);

    // Cleanup
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
