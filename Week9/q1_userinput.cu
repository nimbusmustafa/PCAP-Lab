#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

// CUDA kernel for SpMV
__global__ void spmv_csr_kernel(int num_rows, const float *values, const int *colIndex, const int *rowPtr, const float *x, float *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0.0f;
        int row_start = rowPtr[row];
        int row_end = rowPtr[row + 1];

        for (int j = row_start; j < row_end; j++) {
            dot += values[j] * x[colIndex[j]];
        }
        y[row] = dot;
    }
}

int main() {
    int rows, cols;
    cout << "Enter number of rows and columns: ";
    cin >> rows >> cols;

    // Input dense matrix
    vector<vector<float>> matrix(rows, vector<float>(cols));
    cout << "Enter matrix elements (row-wise):\n";
    for (int i = 0; i < rows; ++i)
        for (int j = 0; j < cols; ++j)
            cin >> matrix[i][j];

    // Input vector
    vector<float> h_x(cols);
    cout << "Enter input vector of size " << cols << ":\n";
    for (int i = 0; i < cols; ++i)
        cin >> h_x[i];

    // Convert dense to CSR format
    vector<float> h_values;
    vector<int> h_colIndex;
    vector<int> h_rowPtr = {0};

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            if (matrix[i][j] != 0) {
                h_values.push_back(matrix[i][j]);
                h_colIndex.push_back(j);
            }
        }
        h_rowPtr.push_back(h_values.size());
    }

    int NNZ = h_values.size(); // Number of non-zeros
    vector<float> h_y(rows);   // Output vector

    // Device pointers
    float *d_values, *d_x, *d_y;
    int *d_colIndex, *d_rowPtr;

    // Allocate memory on GPU
    hipMalloc(&d_values, NNZ * sizeof(float));
    hipMalloc(&d_colIndex, NNZ * sizeof(int));
    hipMalloc(&d_rowPtr, (rows + 1) * sizeof(int));
    hipMalloc(&d_x, cols * sizeof(float));
    hipMalloc(&d_y, rows * sizeof(float));

    // Copy data to GPU
    hipMemcpy(d_values, h_values.data(), NNZ * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_colIndex, h_colIndex.data(), NNZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rowPtr, h_rowPtr.data(), (rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x.data(), cols * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (rows + threadsPerBlock - 1) / threadsPerBlock;
    spmv_csr_kernel<<<blocksPerGrid, threadsPerBlock>>>(rows, d_values, d_colIndex, d_rowPtr, d_x, d_y);

    // Copy result back
    hipMemcpy(h_y.data(), d_y, rows * sizeof(float), hipMemcpyDeviceToHost);

    // Output result
    cout << "Result vector y = [ ";
    for (int i = 0; i < rows; ++i)
        cout << h_y[i] << " ";
    cout << "]\n";

    // Free GPU memory
    hipFree(d_values);
    hipFree(d_colIndex);
    hipFree(d_rowPtr);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}
