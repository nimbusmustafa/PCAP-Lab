#include <iostream>
#include <hip/hip_runtime.h>

#define M 4  // Number of rows of A and C
#define N 5  // Number of columns of B and C
#define K 3  // Number of columns of A and rows of B

__global__ void matrixMultiplyRow(int *A, int *B, int *C, int K1, int N1) {
    int row = blockIdx.x;  // Each thread handles one row of C
    if (row < M) {
        for (int j = 0; j < N1; j++) {
            C[row * N1 + j] = 0;
            for (int k = 0; k < K1; k++) {
                C[row * N1 + j] += A[row * K1 + k] * B[k * N1 + j];
            }
        }
    }
}

int main() {
    int A[M][K] = {{1, 2, 3},
                   {4, 5, 6},
                   {7, 8, 9},
                   {10, 11, 12}};
    int B[K][N] = {{1, 2, 3, 4, 5},
                   {6, 7, 8, 9, 10},
                   {11, 12, 13, 14, 15}};
    int C[M][N];

    int *d_A, *d_B, *d_C;

    // Allocate memory on the device
    hipMalloc((void**)&d_A, M * K * sizeof(int));
    hipMalloc((void**)&d_B, K * N * sizeof(int));
    hipMalloc((void**)&d_C, M * N * sizeof(int));

    // Copy data to device
    hipMemcpy(d_A, A, M * K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with one thread per row
    matrixMultiplyRow<<<M, 1>>>(d_A, d_B, d_C, K, N);

    // Copy result back to host
    hipMemcpy(C, d_C, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Resultant Matrix C:\n";
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
