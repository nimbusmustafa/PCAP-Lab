#include <iostream>
#include <hip/hip_runtime.h>

#define M 4  // Number of rows
#define N 5  // Number of columns

__global__ void matrixAddColumn(int *A, int *B, int *C, int M1 ) {
    int col = blockIdx.x;  // Each thread handles one column
    if (col < N) {
        for (int i = 0; i < M1; i++) {
            C[i * N + col] = A[i * N + col] + B[i * N + col];
        }
    }
}

int main() {
    int A[M][N] = {{1, 2, 3, 4, 5},
                   {6, 7, 8, 9, 10},
                   {11, 12, 13, 14, 15},
                   {16, 17, 18, 19, 20}};
    int B[M][N] = {{21, 22, 23, 24, 25},
                   {26, 27, 28, 29, 30},
                   {31, 32, 33, 34, 35},
                   {36, 37, 38, 39, 40}};
    int C[M][N];

    int *d_A, *d_B, *d_C;

    // Allocate memory on the device
    hipMalloc((void**)&d_A, M * N * sizeof(int));
    hipMalloc((void**)&d_B, M * N * sizeof(int));
    hipMalloc((void**)&d_C, M * N * sizeof(int));

    // Copy data to device
    hipMemcpy(d_A, A, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, M * N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with one thread per column
    matrixAddColumn<<<N, 1>>>(d_A, d_B, d_C, M);

    // Copy result back to host
    hipMemcpy(C, d_C, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Resultant Matrix C:\n";
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
