#include <iostream>
#include <hip/hip_runtime.h>

#define M 4  // Number of rows
#define N 5  // Number of columns

__global__ void matrixAddElement(int *A, int *B, int *C, int M1 , int N1) {
    int row = blockIdx.x;
    int col = threadIdx.x; 
    if (row < M1 && col < N1) {
        C[row * N1 + col] = A[row * N1 + col] + B[row * N1 + col];
    }
}

int main() {
    int A[M][N] = {{1, 2, 3, 4, 5},
                   {6, 7, 8, 9, 10},
                   {11, 12, 13, 14, 15},
                   {16, 17, 18, 19, 20}};
    int B[M][N] = {{21, 22, 23, 24, 25},
                   {26, 27, 28, 29, 30},
                   {31, 32, 33, 34, 35},
                   {36, 37, 38, 39, 40}};
    int C[M][N];

    int *d_A, *d_B, *d_C;

    // Allocate memory on the device
    hipMalloc((void**)&d_A, M * N * sizeof(int));
    hipMalloc((void**)&d_B, M * N * sizeof(int));
    hipMalloc((void**)&d_C, M * N * sizeof(int));

    // Copy data to device
    hipMemcpy(d_A, A, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, M * N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with one thread per element (grid size M, N; block size 1)
    dim3 blockDim(N, 1, 1);
    dim3 gridDim(M, 1, 1);
    matrixAddElement<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N);

    // Copy result back to host
    hipMemcpy(C, d_C, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Resultant Matrix C:\n";
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
